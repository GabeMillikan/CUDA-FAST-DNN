#include "hip/hip_runtime.h"
#include "activation.cuh"

__device__ void Activation::activate(const Activator& activator, const float& in, float* out)
{
	switch (activator)
	{
	default:
	case Activator::Linear:
		*out = in;
		break;
	case Activator::ReLu:
		*out = in <= 0 ? 0 : in;
		break;
	case Activator::Sigmoid:
		*out = 1 / (1 + exp(-in));
		break;
	}
}

void Activation::differentiate(const Activator& activator, const float& in, float* out)
{
	switch (activator)
	{
	default:
	case Activator::Linear:
		*out = in;
		break;
	case Activator::ReLu:
		*out = in <= 0 ? 0 : 1;
		break;
	case Activator::Sigmoid:
		activate(activator, in, out);
		*out *= (1 - *out);
		break;
	}
}
