#include "hip/hip_runtime.h"
#include "network.cuh"
namespace DNN = DeepNeuralNetwork;

DNN::Layer::Layer(size_t height, Activation::Activator activator)
{
	this->height = height;
	this->activator = activator;
}

DNN::Network::Network(
	std::initializer_list<Layer> layers,
	size_t inputHeight,
	size_t trainBatchSize,
	float learningRate,
	float*** weights,
	float** biases
)
{
	int alloc_error = 0;

	this->learningRate = learningRate;
	this->inputHeight = inputHeight;
	this->trainBatchSize = trainBatchSize;
	this->tallestLayerSize = 0; // set during layer loop

	this->layerCount = layers.size();
	
	this->outputHeight = (layers.begin() + this->layerCount - 1)->height;

	//this->shape[layer]
	//this->activators[layer]
	alloc_error |= (int)hipMallocManaged(&this->shape, this->layerCount * sizeof(size_t));
	alloc_error |= (int)hipMallocManaged(&this->activators, this->layerCount * sizeof(Activation::Activator));

	//this->inputs[batch][input_idx]
	//this->expectedOutputs[batch][output_idx]
	//this->unactivatedOutputs[batch][layer][node]
	//this->activatedOutputs[batch][layer][node]
	alloc_error |= (int)hipMallocManaged(&this->inputs, this->trainBatchSize * sizeof(float*));
	alloc_error |= (int)hipMallocManaged(&this->expectedOutputs, this->trainBatchSize * sizeof(float*));
	alloc_error |= (int)hipMallocManaged(&this->unactivatedOutputs, this->trainBatchSize * sizeof(float**));
	alloc_error |= (int)hipMallocManaged(&this->activatedOutputs, this->trainBatchSize * sizeof(float**));
	for (size_t batch = 0; batch < this->trainBatchSize; batch++)
	{
		alloc_error |= (int)hipMallocManaged(this->inputs + batch, this->inputHeight * sizeof(float));
		alloc_error |= (int)hipMallocManaged(this->expectedOutputs + batch, this->outputHeight * sizeof(float));
		alloc_error |= (int)hipMallocManaged(this->unactivatedOutputs + batch, this->layerCount * sizeof(float*));
		alloc_error |= (int)hipMallocManaged(this->activatedOutputs + batch, this->layerCount * sizeof(float*));

		for (size_t i = 0; i < this->layerCount; i++)
		{
			const Layer* layer = layers.begin() + i;
			alloc_error |= (int)hipMallocManaged(this->unactivatedOutputs[batch] + i, layer->height * sizeof(float*));
			alloc_error |= (int)hipMallocManaged(this->activatedOutputs[batch] + i, layer->height * sizeof(float*));
		}
	}

	//this->weights[layer][node][prev_node]
	//this->biases[layer][node]
	alloc_error |= (int)hipMallocManaged(&this->weights, this->layerCount * sizeof(float**));
	alloc_error |= (int)hipMallocManaged(&this->biases, this->layerCount * sizeof(float*));
	size_t previousLayerHeight = inputHeight;
	for (size_t i = 0; i < this->layerCount; i++)
	{
		const Layer* layer = layers.begin() + i;
		this->shape[i] = layer->height;
		this->activators[i] = layer->activator;
		this->tallestLayerSize = layer->height > this->tallestLayerSize ? layer->height : this->tallestLayerSize;

		alloc_error |= (int)hipMallocManaged(this->weights + i, layer->height * sizeof(float*));
		alloc_error |= (int)hipMallocManaged(this->biases + i, layer->height * sizeof(float));

		for (size_t j = 0; j < layer->height; j++)
		{
			alloc_error |= (int)hipMallocManaged(this->weights[i] + j, previousLayerHeight * sizeof(float));

			for (size_t k = 0; k < previousLayerHeight; k++)
				this->weights[i][j][k] = weights ? weights[i][j][k] : randf<-1, 1>() / layer->height;

			this->biases[i][j] = biases ? biases[i][j] : randf<-1, 1>();
		}

		previousLayerHeight = layer->height;
	}

	//this->this_gpuCopy = this (but allocated on the gpu)
	alloc_error |= (int)hipMallocManaged(&this->this_gpuCopy, sizeof(Network));
	*this->this_gpuCopy = *this;

	this->predictionResult = this->activatedOutputs[0][this->layerCount - 1];

	if (alloc_error)
	{
		printf("There was at least one error while allocating memory. The bit mask of all errors is: %d\n", alloc_error);
		exit(1);
	}
}

void DNN::Network::predict(float* inputs)
{
	memcpy(this->inputs[0], inputs, this->inputHeight * sizeof(float));
	DNN::Utils::feedForward<<<1, this->tallestLayerSize>>>(this->this_gpuCopy);
	hipDeviceSynchronize();
}

void DNN::Network::train(float** inputs, float** outputs)
{
	for (size_t batch = 0; batch < this->trainBatchSize; batch++)
	{
		memcpy(this->inputs[batch], inputs[batch], this->inputHeight * sizeof(float));
		memcpy(this->expectedOutputs[batch], outputs[batch], this->outputHeight * sizeof(float));
	}

	DNN::Utils::feedForward<<<this->trainBatchSize, this->tallestLayerSize>>>(this->this_gpuCopy);
	hipDeviceSynchronize();

	DNN::Utils::backPropRecord<<<this->trainBatchSize, this->tallestLayerSize>>>(this->this_gpuCopy);
	hipDeviceSynchronize();
	
	DNN::Utils::backPropUpdate<<<this->layerCount, this->tallestLayerSize>>>(this->this_gpuCopy);
	hipDeviceSynchronize();
}

DNN::Network::~Network()
{
	//this->this_gpuCopy = this (but allocated on the gpu)
	hipFree(this->this_gpuCopy);

	//this->biases[layer][node]
	//this->weights[layer][node][prev_node]
	for (size_t layer = 0; layer < this->layerCount; layer++)
	{
		for (size_t node = 0; node < this->shape[node]; node++)
		{
			hipFree(this->weights[layer][node]);
		}
		hipFree(this->biases[layer]);
		hipFree(this->weights[layer]);
	}
	hipFree(this->biases);
	hipFree(this->weights);

	//this->activatedOutputs[batch][layer][node]
	//this->unactivatedOutputs[batch][layer][node]
	//this->expectedOutputs[batch][output_idx]
	//this->inputs[batch][input_idx]
	for (size_t batch = 0; batch < this->trainBatchSize; batch++)
	{
		for (size_t layer = 0; layer < this->layerCount; layer++)
		{
			hipFree(this->activatedOutputs[batch][layer]);
			hipFree(this->unactivatedOutputs[batch][layer]);
		}
		hipFree(this->activatedOutputs[batch]);
		hipFree(this->unactivatedOutputs[batch]);
		hipFree(this->expectedOutputs[batch]);
		hipFree(this->inputs[batch]);
	}
	hipFree(this->activatedOutputs);
	hipFree(this->unactivatedOutputs);
	hipFree(this->expectedOutputs);
	hipFree(this->inputs);

	//this->activators[layer]
	//this->shape[layer]
	hipFree(this->activators);
	hipFree(this->shape);
}

__global__ void DNN::Utils::feedForward(Network* nn)
{
	const size_t& batch = blockIdx.x;
	const size_t& node = threadIdx.x;
	size_t layer = 0;

	while (layer < nn->layerCount)
	{
		// perform feedforward
		if (node < nn->shape[layer])
		{
			const Activation::Activator& activator = nn->activators[layer];
			
			float* unactivatedOutput = nn->unactivatedOutputs[batch][layer] + node;
			*unactivatedOutput = 0;

			if (layer == 0)
			{
				for (size_t j = 0; j < nn->inputHeight; j++)
				{
					*unactivatedOutput += nn->inputs[batch][j] * nn->weights[0][node][j] + nn->biases[0][node];
				}
			}
			else
			{
				const size_t& prevLayer = layer - 1;
				const size_t& prevLayerHeight = nn->shape[prevLayer];
				for (size_t j = 0; j < prevLayerHeight; j++)
				{
					*unactivatedOutput += nn->activatedOutputs[batch][prevLayer][j] * nn->weights[layer][node][j] + nn->biases[layer][node];
				}
			}

			Activation::activate(activator, *unactivatedOutput, nn->activatedOutputs[batch][layer] + node);
		}
		
		// sync everything up and move on to next layer
		__syncthreads();
		layer++;
	}
}

__global__ void DNN::Utils::backPropRecord(Network* nn)
{
	/*
	These equations are followed religiously:
	https://cdn.discordapp.com/attachments/282239317966061568/902381475549282354/342191494872039424.png

	o_{xi} = \sum_{j=1}^{I_{x-1}}w_{xij}a_{(x-1)j}+b_{xi} \\
	a_{xi} = \sigma(o_{xi}) \\
	C = \frac{1}{I}\sum_{i=1}^{I_X}(a_{Xi}-t_i)^2 \\
	\frac{dC}{do_{Xi}} = \frac{2}{I_X}(a_{Xi} - t_{i})\sigma ^ \prime (o_{Xi}) \\
	\frac{dC}{do_{xi}} = \sum_{j=1}^{I_{x+1}}\frac{dC}{do_{(x+1)j}} \cdot w_{(x+1)ji} \cdot \sigma ^\prime (o_{xi}) \\
	\frac{do_{xi}}{dw_{xij}} = a_{(x-1)j} \\
	\frac{do_{xi}}{db_{xi}} = 1 \\
	\frac{dC}{dw_{xij}} = \frac{dC}{do_{xi}}\cdot \frac{do_{xi}}{dw_{xij}} = \frac{dC}{do_{xi}}\cdot a_{(x-1)j}\\
	\frac{dC}{db_{xi}} = \frac{dC}{do_{xi}}\cdot \frac{do_{xi}}{db_{xi}} = \frac{dC}{do_{xi}}

	nn->unactivatedOutputs will be used to store dc/do
	*/

	const size_t& batch = blockIdx.x;
	const size_t& node = threadIdx.x;

	// calculate dc/do for the very last layer
	size_t layer = nn->layerCount - 1;
	if (node < nn->outputHeight)
	{
		float* o_Xi = nn->unactivatedOutputs[batch][layer] + node;
		const float err = *o_Xi - nn->expectedOutputs[batch][node];

		Activation::differentiate(nn->activators[layer], *o_Xi, o_Xi);
		*o_Xi *= 2.f * err / nn->outputHeight;
	}

	// now for every other layer
	--layer;
	size_t layerHeight = nn->shape[layer], followingLayerHeight = nn->outputHeight, j = 0;
	while (layer != std::numeric_limits<size_t>::max())
	{
		__syncthreads();
		if (node < layerHeight)
		{
			float* o_xi = nn->unactivatedOutputs[batch][layer] + node;
			Activation::differentiate(nn->activators[layer], *o_xi, o_xi);

			++layer;
			float j_sum = 0.f;
			for (j = 0; j < followingLayerHeight; ++j)
				j_sum += nn->unactivatedOutputs[batch][layer][j] * nn->weights[layer][j][node];
			--layer;

			*o_xi *= j_sum;
			//*a_xi *= j_sum / followingLayerHeight; // divide just to normalize into reasonable range
		}

		--layer;
		followingLayerHeight = layerHeight;
		layerHeight = nn->shape[layer];
	}

	// now, all of the dc/do information is stored in nn->activatedOutputs
	// next: call backPropUpdate()
}


void DNN::Utils::backPropUpdate(Network* nn)
{
	const size_t& layer = blockIdx.x;
	const size_t& node = threadIdx.x;
	const size_t& batchSize = nn->trainBatchSize;
	const size_t& prevLayerHeight = layer == 0 ? nn->inputHeight : nn->shape[layer - 1];

	if (nn->shape[layer] <= node) return;

	float avg_dc_do = 0.f;
	for (size_t batch = 0; batch < batchSize; ++batch)
	{
		avg_dc_do += nn->activatedOutputs[batch][layer][node];
	}
	avg_dc_do /= batchSize;

	// weights
	for (size_t weight = 0; weight < prevLayerHeight; weight++)
	{
		nn->weights[layer][node][weight] -= avg_dc_do * weight * nn->learningRate;
	}
}