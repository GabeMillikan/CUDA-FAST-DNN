#include "hip/hip_runtime.h"
#include "network.cuh"
namespace DNN = DeepNeuralNetwork;

DNN::Layer::Layer(size_t height, Activation::Activator activator)
{
	this->height = height;
	this->activator = activator;
}

DNN::Network::Network(
	std::initializer_list<Layer> layers,
	size_t inputHeight,
	size_t trainBatchSize,
	float learningRate,
	float*** weights,
	float** biases
)
{
	int alloc_error = 0;

	this->learningRate = learningRate;
	this->inputHeight = inputHeight;
	this->trainBatchSize = trainBatchSize;
	this->tallestLayerSize = 0; // set during layer loop

	this->layerCount = layers.size();
	
	this->outputHeight = (layers.begin() + this->layerCount - 1)->height;

	//this->shape[layer]
	//this->activators[layer]
	alloc_error |= (int)hipMallocManaged(&this->shape, this->layerCount * sizeof(size_t));
	alloc_error |= (int)hipMallocManaged(&this->activators, this->layerCount * sizeof(Activation::Activator));

	//this->inputs[batch][input_idx]
	//this->expectedOutputs[batch][output_idx]
	//this->unactivatedOutputs[batch][layer][node]
	//this->activatedOutputs[batch][layer][node]
	alloc_error |= (int)hipMallocManaged(&this->inputs, this->trainBatchSize * sizeof(float*));
	alloc_error |= (int)hipMallocManaged(&this->expectedOutputs, this->trainBatchSize * sizeof(float*));
	alloc_error |= (int)hipMallocManaged(&this->unactivatedOutputs, this->trainBatchSize * sizeof(float**));
	alloc_error |= (int)hipMallocManaged(&this->activatedOutputs, this->trainBatchSize * sizeof(float**));
	for (size_t batch = 0; batch < this->trainBatchSize; batch++)
	{
		alloc_error |= (int)hipMallocManaged(this->inputs + batch, this->inputHeight * sizeof(float));
		alloc_error |= (int)hipMallocManaged(this->expectedOutputs + batch, this->outputHeight * sizeof(float));
		alloc_error |= (int)hipMallocManaged(this->unactivatedOutputs + batch, this->layerCount * sizeof(float*));
		alloc_error |= (int)hipMallocManaged(this->activatedOutputs + batch, this->layerCount * sizeof(float*));

		for (size_t i = 0; i < this->layerCount; i++)
		{
			const Layer* layer = layers.begin() + i;
			alloc_error |= (int)hipMallocManaged(this->unactivatedOutputs[batch] + i, layer->height * sizeof(float*));
			alloc_error |= (int)hipMallocManaged(this->activatedOutputs[batch] + i, layer->height * sizeof(float*));
		}
	}

	//this->weights[layer][node][prev_node]
	//this->biases[layer][node]
	alloc_error |= (int)hipMallocManaged(&this->weights, this->layerCount * sizeof(float**));
	alloc_error |= (int)hipMallocManaged(&this->biases, this->layerCount * sizeof(float*));
	size_t previousLayerHeight = inputHeight;
	for (size_t i = 0; i < this->layerCount; i++)
	{
		const Layer* layer = layers.begin() + i;
		this->shape[i] = layer->height;
		this->activators[i] = layer->activator;
		this->tallestLayerSize = layer->height > this->tallestLayerSize ? layer->height : this->tallestLayerSize;

		alloc_error |= (int)hipMallocManaged(this->weights + i, layer->height * sizeof(float*));
		alloc_error |= (int)hipMallocManaged(this->biases + i, layer->height * sizeof(float));

		for (size_t j = 0; j < layer->height; j++)
		{
			alloc_error |= (int)hipMallocManaged(this->weights[i] + j, previousLayerHeight * sizeof(float));

			for (size_t k = 0; k < previousLayerHeight; k++)
				this->weights[i][j][k] = weights ? weights[i][j][k] : randf<-1, 1>() / layer->height;

			this->biases[i][j] = biases ? biases[i][j] : randf<-1, 1>();
		}

		previousLayerHeight = layer->height;
	}

	//this->this_gpuCopy = this (but allocated on the gpu)
	alloc_error |= (int)hipMallocManaged(&this->this_gpuCopy, sizeof(Network));
	*this->this_gpuCopy = *this;

	this->predictionResult = this->activatedOutputs[0][this->layerCount - 1];

	if (alloc_error)
	{
		printf("There was at least one error while allocating memory. The bit mask of all errors is: %d\n", alloc_error);
		exit(1);
	}
}

void DNN::Network::predict(float* inputs)
{
	memcpy(this->inputs[0], inputs, this->inputHeight * sizeof(float));
	DNN::Utils::feedForward<<<1u, (unsigned int)this->tallestLayerSize>>>(this->this_gpuCopy);
	hipDeviceSynchronize();
}

void DNN::Network::train(float** inputs, float** outputs)
{
	for (size_t batch = 0; batch < this->trainBatchSize; batch++)
	{
		memcpy(this->inputs[batch], inputs[batch], this->inputHeight * sizeof(float));
		memcpy(this->expectedOutputs[batch], outputs[batch], this->outputHeight * sizeof(float));
	}

	DNN::Utils::feedForward<<<(unsigned int)this->trainBatchSize, (unsigned int)this->tallestLayerSize>>>(this->this_gpuCopy);
	hipDeviceSynchronize();

	DNN::Utils::backPropRecord<<<(unsigned int)this->trainBatchSize, (unsigned int)this->tallestLayerSize>>>(this->this_gpuCopy);
	hipDeviceSynchronize();
	
	DNN::Utils::backPropUpdate<<<(unsigned int)this->layerCount, (unsigned int)this->tallestLayerSize>>>(this->this_gpuCopy);
	hipDeviceSynchronize();
}

void DeepNeuralNetwork::Network::summary(bool showParameterValues)
{
	printf("================ NETWORK SUMMARY ================\n");
	printf("    Inputs: %5d             Outputs: %5d\n", (int)this->inputHeight, (int)this->outputHeight);
	printf("    Layers: %5d          Batch Size: %5d\n", (int)this->layerCount, (int)this->trainBatchSize);
	printf("            Learning Rate: %.4g\n", this->learningRate);
	printf("================     LAYERS     =================\n");
	for (size_t layer = 0; layer < this->layerCount; layer++)
	{
		const Activation::Activator& activator = this->activators[layer];
		const size_t& height = this->shape[layer];

		printf(" %4d.) %d neuron(s), %s activation\n", (int)layer, (int)height, Activation::stringifyActivator(activator));
		if (showParameterValues)
		{
			printf("      Biases: [");
			for (size_t node = 0; node < height; node++)
			{
				printf(node == 0 ? "%.3f" : ", %.3f", this->biases[layer][node]);
			}

			const size_t& prevHeight = layer == 0 ? this->inputHeight : this->shape[layer - 1];
			printf("]\n      Weights: \n");
			for (size_t node = 0; node < height; node++)
			{
				printf("          [");
				for (size_t j = 0; j < prevHeight; j++)
				{
					printf(j == 0 ? "%.3f" : ", %.3f", this->weights[layer][node][j]);
				}
				printf("]\n");
			}
			printf("      ]\n");
		}
	}
	printf("=================================================\n");

	printf("\n\n");
}

DNN::Network::~Network()
{
	//this->this_gpuCopy = this (but allocated on the gpu)
	hipFree(this->this_gpuCopy);

	//this->biases[layer][node]
	//this->weights[layer][node][prev_node]
	for (size_t layer = 0; layer < this->layerCount; layer++)
	{
		for (size_t node = 0; node < this->shape[node]; node++)
		{
			hipFree(this->weights[layer][node]);
		}
		hipFree(this->biases[layer]);
		hipFree(this->weights[layer]);
	}
	hipFree(this->biases);
	hipFree(this->weights);

	//this->activatedOutputs[batch][layer][node]
	//this->unactivatedOutputs[batch][layer][node]
	//this->expectedOutputs[batch][output_idx]
	//this->inputs[batch][input_idx]
	for (size_t batch = 0; batch < this->trainBatchSize; batch++)
	{
		for (size_t layer = 0; layer < this->layerCount; layer++)
		{
			hipFree(this->activatedOutputs[batch][layer]);
			hipFree(this->unactivatedOutputs[batch][layer]);
		}
		hipFree(this->activatedOutputs[batch]);
		hipFree(this->unactivatedOutputs[batch]);
		hipFree(this->expectedOutputs[batch]);
		hipFree(this->inputs[batch]);
	}
	hipFree(this->activatedOutputs);
	hipFree(this->unactivatedOutputs);
	hipFree(this->expectedOutputs);
	hipFree(this->inputs);

	//this->activators[layer]
	//this->shape[layer]
	hipFree(this->activators);
	hipFree(this->shape);
}

__global__ void DNN::Utils::feedForward(Network* nn)
{
	const size_t& batch = blockIdx.x;
	const size_t& node = threadIdx.x;
	size_t layer = 0;
	printf("FF thread <%d, %d>\n", (int)batch, (int)node);

	while (layer < nn->layerCount)
	{
		// perform feedforward
		if (node < nn->shape[layer])
		{
			const Activation::Activator& activator = nn->activators[layer];
			
			float* unactivatedOutput = nn->unactivatedOutputs[batch][layer] + node;
			*unactivatedOutput = nn->biases[layer][node];
			printf("unactivatedOutput[%d][%d][%d] = %.3f\n", (int)batch, (int)layer, (int)node, *unactivatedOutput);

			if (layer == 0)
			{
				for (size_t j = 0; j < nn->inputHeight; j++)
				{
					*unactivatedOutput += nn->inputs[batch][j] * nn->weights[0][node][j];
					printf("unactivatedOutput[%d][%d][%d] += %.3f * %.3f\n", (int)batch, (int)layer, (int)node, nn->inputs[batch][j], nn->weights[0][node][j]);
				}
			}
			else
			{
				const size_t& prevLayer = layer - 1;
				const size_t& prevLayerHeight = nn->shape[prevLayer];
				for (size_t j = 0; j < prevLayerHeight; j++)
				{
					*unactivatedOutput += nn->activatedOutputs[batch][prevLayer][j] * nn->weights[layer][node][j];
					printf("unactivatedOutput[%d][%d][%d] += %.3f * %.3f\n", (int)batch, (int)layer, (int)node, nn->activatedOutputs[batch][prevLayer][j], nn->weights[0][node][j]);
				}
			}

			printf("unactivatedOutput[%d][%d][%d] ==> %.3f\n", (int)batch, (int)layer, (int)node, *unactivatedOutput);
			Activation::activate(activator, *unactivatedOutput, nn->activatedOutputs[batch][layer] + node);
			printf("activatedOutputs[%d][%d][%d] = %.3f\n", (int)batch, (int)layer, (int)node, nn->activatedOutputs[batch][layer][node]);
		}
		
		// sync everything up and move on to next layer
		__syncthreads();
		layer++;
	}
}

__global__ void DNN::Utils::backPropRecord(Network* nn)
{
	/*
	These equations are followed religiously:
	https://cdn.discordapp.com/attachments/282239317966061568/902381475549282354/342191494872039424.png

	o_{xi} = \sum_{j=1}^{I_{x-1}}w_{xij}a_{(x-1)j}+b_{xi} \\
	a_{xi} = \sigma(o_{xi}) \\
	C = \frac{1}{I}\sum_{i=1}^{I_X}(a_{Xi}-t_i)^2 \\
	\frac{dC}{do_{Xi}} = \frac{2}{I_X}(a_{Xi} - t_{i})\sigma ^ \prime (o_{Xi}) \\
	\frac{dC}{do_{xi}} = \sum_{j=1}^{I_{x+1}}\frac{dC}{do_{(x+1)j}} \cdot w_{(x+1)ji} \cdot \sigma ^\prime (o_{xi}) \\
	\frac{do_{xi}}{dw_{xij}} = a_{(x-1)j} \\
	\frac{do_{xi}}{db_{xi}} = 1 \\
	\frac{dC}{dw_{xij}} = \frac{dC}{do_{xi}}\cdot \frac{do_{xi}}{dw_{xij}} = \frac{dC}{do_{xi}}\cdot a_{(x-1)j}\\
	\frac{dC}{db_{xi}} = \frac{dC}{do_{xi}}\cdot \frac{do_{xi}}{db_{xi}} = \frac{dC}{do_{xi}}

	nn->unactivatedOutputs will be used to store dc/do
	*/

	const size_t& batch = blockIdx.x;
	const size_t& node = threadIdx.x;
	printf("BPR thread <%d, %d>\n", (int)batch, (int)node);

	// calculate dc/do for the very last layer
	size_t layer = nn->layerCount - 1;
	if (node < nn->outputHeight)
	{
		float* o_Xi = nn->unactivatedOutputs[batch][layer] + node;
		Activation::differentiate(nn->activators[layer], *o_Xi, o_Xi);
		*o_Xi *= 2.f * (nn->activatedOutputs[batch][layer][node] - nn->expectedOutputs[batch][node]) / nn->outputHeight;
		printf("dc/do batch,layer,node=%d,%d,%d = %.3f\n", (int)batch, (int)layer, (int)node, *o_Xi);
	}

	// now for every other layer
	--layer;
	size_t layerHeight = nn->shape[layer], followingLayerHeight = nn->outputHeight, j = 0;
	while (layer != (size_t)-1)
	{
		__syncthreads();
		if (node < layerHeight)
		{
			float* o_xi = nn->unactivatedOutputs[batch][layer] + node;
			Activation::differentiate(nn->activators[layer], *o_xi, o_xi);

			++layer;
			float j_sum = 0.f;
			for (j = 0; j < followingLayerHeight; ++j)
				j_sum += nn->unactivatedOutputs[batch][layer][j] * nn->weights[layer][j][node];
			--layer;

			*o_xi *= j_sum;
			//*a_xi *= j_sum / followingLayerHeight; // divide just to normalize into reasonable range
		}

		--layer;
		followingLayerHeight = layerHeight;
		layerHeight = nn->shape[layer];
	}

	// now, all of the dc/do information is stored in nn->activatedOutputs
	// next: call backPropUpdate()
}


__global__ void DNN::Utils::backPropUpdate(Network* nn)
{
	const size_t& layer = blockIdx.x;
	const size_t& node = threadIdx.x;
	const size_t& batchSize = nn->trainBatchSize;
	const size_t& prevLayerHeight = layer == 0 ? nn->inputHeight : nn->shape[layer - 1];

	printf("BPU thread <%d, %d>\n", (int)layer, (int)node);
	if (nn->shape[layer] <= node) return;

	// dc/db = dc/do
	float dc_dx = 0.f;
	for (size_t batch = 0; batch < batchSize; ++batch)
	{
		dc_dx += nn->unactivatedOutputs[batch][layer][node];
	}
	dc_dx /= batchSize;

	// gradient descent 
	nn->biases[layer][node] -= dc_dx * nn->learningRate;

	// dc/dw = dc/do * prev activated
	for (size_t weight = 0; weight < prevLayerHeight; weight++)
	{
		dc_dx = 0.f;
		if (layer == 0)
		{
			for (size_t batch = 0; batch < batchSize; ++batch)
			{
				dc_dx += nn->unactivatedOutputs[batch][layer][node] * nn->inputs[batch][weight];
			}
		}
		else
		{
			for (size_t batch = 0; batch < batchSize; ++batch)
			{
				dc_dx += nn->unactivatedOutputs[batch][layer][node] * nn->activatedOutputs[batch][layer - 1][weight];
			}
		}
		dc_dx /= batchSize;

		// gradient descent 
		nn->weights[layer][node][weight] -= dc_dx * nn->learningRate;
	}
}